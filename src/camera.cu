#include "hip/hip_runtime.h"
#include "camera.cuh"

Camera::Camera(float vfov, float aspect, const Vec3 &look_from, const Vec3 &look_at, const Vec3 &view_up) {
  pos = look_from;
  Vec3 u, v, w;

  float half_height = tanf(vfov / 2); // vfov should be in radians
  float half_width = half_height * aspect;

  w = normalized(look_from - look_at);
  u = normalized(cross(view_up, w));
  v = cross(w, u);

  lower_left = look_from - (half_width * u) - (half_height * v) - w;

  h = 2 * half_width * u;
  v = 2 * half_height * v;
}

Ray Camera::get_ray(float s, float t) const {
  return { pos, lower_left + (s * h) + (t * v) - pos };
}

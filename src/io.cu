#include "hip/hip_runtime.h"
#include "io.cuh"
#include "bxdf.cuh"
#include "fresnel.cuh"
#include "bsdf.cuh"

#include <iostream>
#include <fstream>
#include <cstdio>
#include <vector>
#include <map>

std::string ltrim(const std::string &s) {
  size_t start = s.find_first_not_of(" \n\r\t");
  return (start == std::string::npos) ? "" : s.substr(start);
}

std::string rtrim(const std::string &s) {
  size_t end = s.find_last_not_of(" \n\r\t");
  return (end == std::string::npos) ? "" : s.substr(0, end + 1);
}

std::string trim(const std::string &s) {
  return rtrim(ltrim(s));
}

void load_material(std::map<std::string, BSDF> &materials, const std::string &name, float Ns, const Vec3 &Kd, const Vec3 &Ks, const Vec3 &Ke) {
  if (name != "") {
    if (!is_zero(Ke)) {
      materials[name] = BSDF(new AreaLight(Kd, Ke));
    } else if (is_zero(Ks)) {
      materials[name] = BSDF(new Lambertian(Kd));
    } else {
      float roughness = 1.f - sqrtf(Ns) / 30.f;
      materials[name] = BSDF(new TorranceSparrow(Kd, new Beckmann(roughness), new Fresnel(1.0f, 1.5f)));
    }
  }
}

void load_materials(std::string &fname, std::map<std::string, BSDF> &materials) {
  std::string current_name = "";
  Vec3 Kd, Ks, Ke;
  float Ns;

  std::ifstream input(fname);
  std::string line;
  for (; std::getline(input, line); ) {
    line = trim(line);
    if (line.find("newmtl") != std::string::npos) {
      load_material(materials, current_name, Ns, Kd, Ks, Ke);
      current_name = line.substr(7);
      Kd = {0.f, 0.f, 0.f};
      Ks = {0.f, 0.f, 0.f};
      Ke = {0.f, 0.f, 0.f};
      Ns = 0.f;
    } else if (line[0] == 'K' && line[1] == 'd') {
      sscanf(line.c_str(), "Kd %f %f %f", Kd.e, Kd.e + 1, Kd.e + 2);
    } else if (line[0] == 'K' && line[1] == 's') {
      sscanf(line.c_str(), "Ks %f %f %f", Ks.e, Ks.e + 1, Ks.e + 2);
    } else if (line[0] == 'K' && line[1] == 'e') {
      sscanf(line.c_str(), "Ke %f %f %f", Ke.e, Ke.e + 1, Ke.e + 2);
    } else if (line[0] == 'N' && line[1] == 's') {
      sscanf(line.c_str(), "Ns %f", &Ns);
    }
  }
  load_material(materials, current_name, Ns, Kd, Ks, Ke);
}

void load_vertex(const std::string &line, std::vector<Vec3> &verts) {
  float x, y, z;
  sscanf(line.c_str(), "v %f %f %f", &x, &y, &z);
  verts.emplace_back(x, y, z);
}

void load_normal(const std::string &line, std::vector<Vec3> &normals) {
  float x, y, z;
  sscanf(line.c_str(), "vn %f %f %f", &x, &y, &z);
  normals.emplace_back(x, y, z);
}

void clean(int e[9], int n) {
  for (int i = 0; i < 9; i++) {
    e[i] = (e[i] < 0 ? n + e[i] : e[i] - 1);
  }
}

void clean_alt(int e[9], int n1, int n2) {
  for (int i = 0; i < 9; i++) {
    e[i] = (e[i] < 0 ? ((i % 2 == 0) ? n1 : n2) + e[i] : e[i] - 1);
  }
}

void load_face(const std::string &line, std::string &current_name, const std::vector<Vec3> &verts, const std::vector<Vec3> &normals, const std::map<std::string, BSDF> materials, BSDF *mat_arr, std::vector<Primitive> &prims) {
  int mat_idx = std::distance(materials.begin(), materials.find(current_name));
  if (mat_idx == materials.size()) {
    printf("No material with name %s\n", current_name.c_str());
    current_name = "";
    return load_face(line, current_name, verts, normals, materials, mat_arr, prims);
  }
  BSDF *bsdf = mat_arr + mat_idx;

  int n;
  int e[9];

  n = sscanf(line.c_str(), "f %d %d %d %d", e+0, e+1, e+2, e+3);
  if (n >= 3) {
    clean(e, verts.size());
    prims.emplace_back(Tri(verts[e[0]], verts[e[1]], verts[e[2]]), bsdf);
    if (n == 4) prims.emplace_back(Tri(verts[e[1]], verts[e[2]], verts[e[3]]), bsdf);
    return;
  }

  n = sscanf(line.c_str(), "f %d/%*d %d/%*d %d/%*d %d/%*d", e+0, e+1, e+2, e+3);
  if (n >= 3) {
    clean(e, verts.size());
    prims.emplace_back(Tri(verts[e[0]], verts[e[1]], verts[e[2]]), bsdf);
    if (n == 4) prims.emplace_back(Tri(verts[e[1]], verts[e[2]], verts[e[3]]), bsdf);
    return;
  }

  n = sscanf(line.c_str(), "f %d/%*d/%d %d/%*d/%d %d/%*d/%d %d/%*d/%d", e+0, e+1, e+2, e+3, e+4, e+5, e+6, e+7);
  if (n >= 6) {
    clean_alt(e, verts.size(), normals.size());
    prims.emplace_back(Tri(verts[e[0]], verts[e[2]], verts[e[4]], normals[e[1]], normals[e[3]], normals[e[5]]), bsdf);
    if (n == 8) prims.emplace_back(Tri(verts[e[2]], verts[e[4]], verts[e[6]], normals[e[3]], normals[e[5]], normals[e[7]]), bsdf);
    return;
  }

  n = sscanf(line.c_str(), "f %d//%d %d//%d %d//%d %d//%d", e+0, e+1, e+2, e+3, e+4, e+5, e+6, e+7);
  if (n >= 6) {
    clean_alt(e, verts.size(), normals.size());
    prims.emplace_back(Tri(verts[e[0]], verts[e[2]], verts[e[4]], normals[e[1]], normals[e[3]], normals[e[5]]), bsdf);
    if (n > 6) prims.emplace_back(Tri(verts[e[2]], verts[e[4]], verts[e[6]], normals[e[3]], normals[e[5]], normals[e[7]]), bsdf);
    return;
  }

  printf("Cannot parse line: %s\n", line.c_str());
  exit(1);
}

Scene load_obj(std::string fname) {
  Camera cam;

  std::ifstream input(fname);
  std::map<std::string, BSDF> materials;
  materials[""] = BSDF(new Lambertian(Vec3(1.f, 1.f, 1.f)));
  fname.replace(fname.end() - 3, fname.end(), "mtl");
  std::string current_name = "";
  load_materials(fname, materials);

  int n_mats = materials.size();
  BSDF *mats = (BSDF *)malloc(n_mats * sizeof(BSDF));
  int i = 0;
  for (auto it = materials.begin(); it != materials.end(); it++, i++) mats[i] = it->second; 

  std::vector<Vec3> verts;
  std::vector<Vec3> normals;
  std::vector<Primitive> prims;

  for (std::string line; std::getline(input, line); ) {
    line = trim(line);
    if (line[0] == '#' || line == "" || line == "\n" || line[0] == 'g' || line[0] == 'o' || line[0] == 's') {
      continue;
    } else if (line[0] == 'v' && line[1] == ' ') {
      load_vertex(line, verts);
    } else if (line[0] == 'v' && line[1] == 'n') {
      load_normal(line, normals);
    } else if (line.find("usemtl") != std::string::npos) {
      current_name = line.substr(7);
    } else if (line[0] == 'f') {
      load_face(line, current_name, verts, normals, materials, mats, prims);
    } else {
      printf("Unrecognized line %s\n", line.c_str());
    }
  }
  Primitive *prim_arr = (Primitive *)malloc(prims.size() * sizeof(Primitive));

  int n_lights = 0;
  for (int i = 0; i < prims.size(); i++) {
    prim_arr[i] = prims[i];
    if (prims[i].bsdf->is_light()) n_lights++;
  }

  BVH bvh = build_bvh(prim_arr, prims.size());

  int light = 0;
  Primitive **lights = (Primitive **)malloc(n_lights * sizeof(Primitive *));
  for (int i = 0; i < prims.size(); i++) {
    if (prim_arr[i].bsdf->is_light()) lights[light++] = prim_arr + i;
  }

  return {cam, bvh, lights, n_lights, mats, n_mats};
}

void write_ppm(const std::string &fname, const Image &im) {
  std::ofstream output(fname);

  output << "P3\n" << im.width << ' ' << im.height << "\n255\n";

  Vec3 rgb;
  int r, g, b;
  for (int j = im.height - 1; j >= 0; j--) {
    for (int i = 0; i < im.width; i++) {
      rgb = im.film[j * im.width + i];
      rgb.e[0] = (std::isnan(rgb.e[0])) ? 0.f : ((rgb.e[0] > 1.f) ? 1.f : rgb.e[0]);
      rgb.e[1] = (std::isnan(rgb.e[1])) ? 0.f : ((rgb.e[1] > 1.f) ? 1.f : rgb.e[1]);
      rgb.e[2] = (std::isnan(rgb.e[2])) ? 0.f : ((rgb.e[2] > 1.f) ? 1.f : rgb.e[2]);
      r = (int)(255.999 * sqrtf(rgb.e[0]));
      g = (int)(255.999 * sqrtf(rgb.e[1]));
      b = (int)(255.999 * sqrtf(rgb.e[2]));
      output << r << ' ' << g << ' ' << b << '\n';
    }
  }
}

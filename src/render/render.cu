#include "hip/hip_runtime.h"
#include "camera.cuh"
#include "image.cuh"
#include "render.cuh"
#include "sphere.cuh"
#include "trimesh.cuh"

#include <hip/hip_runtime.h>
#include <iostream>

#define cudaCheckError()                                                               \
  {                                                                                    \
    hipError_t e = hipGetLastError();                                                \
    if (e != hipSuccess) {                                                            \
      printf("Cuda failure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e)); \
      exit(0);                                                                         \
    }                                                                                  \
  }

__global__ void render_kernel_normals(TriMesh m, Camera cam, float3* out, unsigned int w, unsigned int h) {
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;

  if (x >= w || y >= h)
    return;

  int spp_x  = 4;
  int spp_y  = 4;
  int spp    = spp_x * spp_y;
  float3 rgb = {0, 0, 0};

  for (int xx = 0; xx < spp_x; xx++) {
    for (int yy = 0; yy < spp_y; yy++) {
      float u = ((float) x + (float) xx / (float) spp_x) / (float) w;
      float v = ((float) y + (float) yy / (float) spp_y) / (float) h;

      Ray r = cam.get_ray(u, v);

      auto i   = m.intersects(r);
      auto tri = i.tri;

      auto normals = TriangleNormals(tri);
      Vec3 normal  = normals.at(i.uvw, r);
      if (i.hit) {
        rgb.x += (normal.x + 1.0) / 2.0;
        rgb.y += (normal.y + 1.0) / 2.0;
        rgb.z += (normal.z + 1.0) / 2.0;
      }
    }
  }

  out[y * w + x] = rgb / (float) spp;
}

Image render_normals(TriMesh host_mesh, Camera cam, unsigned int w, unsigned int h) {
  Image out(w, h);

  float3* device_out;
  hipMalloc(&device_out, w * h * sizeof(float3));
  cudaCheckError();

  Triangle* device_tris;
  hipMalloc(&device_tris, host_mesh.n * sizeof(Triangle));
  hipMemcpy(device_tris, host_mesh.tris, host_mesh.n * sizeof(Triangle), hipMemcpyHostToDevice);
  cudaCheckError();
  TriMesh device_mesh(device_tris, host_mesh.n);

  dim3 block(16, 16);
  dim3 grid((w + 15) / 16, (h + 15) / 16);
  render_kernel_normals<<<grid, block>>>(device_mesh, cam, device_out, w, h);
  cudaCheckError();

  hipDeviceSynchronize();
  hipMemcpy(out.data, device_out, w * h * sizeof(float3), hipMemcpyDeviceToHost);
  cudaCheckError();
  hipFree(device_out);
  hipFree(device_tris);

  return out;
}

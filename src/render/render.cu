#include "hip/hip_runtime.h"
#include "camera.cuh"
#include "image.cuh"
#include "render.cuh"
#include "sphere.cuh"
#include "trimesh.cuh"

#include <hip/hip_runtime.h>
#include <iostream>

#define cudaCheckError()                                                               \
  {                                                                                    \
    hipError_t e = hipGetLastError();                                                \
    if (e != hipSuccess) {                                                            \
      printf("Cuda failure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e)); \
      exit(0);                                                                         \
    }                                                                                  \
  }

__global__ void render_kernel_normals(Triangle t, Camera cam, float3* out, unsigned int w, unsigned int h) {
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;

  if (x >= w || y >= h)
    return;

  int spp_x  = 20;
  int spp_y  = 20;
  int spp    = spp_x * spp_y;
  float3 rgb = {0, 0, 0};

  for (int xx = 0; xx < spp_x; xx++) {
    for (int yy = 0; yy < spp_y; yy++) {
      float u = ((float) x + (float) xx / (float) spp_x) / (float) w;
      float v = ((float) y + (float) yy / (float) spp_y) / (float) h;

      Ray r = cam.get_ray(u, v);

      auto i = t.intersects(r);

      auto normals = TriangleNormals(t);
      rgb += i.uvw;
      // Vec3 normal  = normals.at(i.uvw, r);
      // if (i.hit) {
      //   rgb.x += (normal.x + 1.0) / 2.0;
      //   rgb.y += (normal.y + 1.0) / 2.0;
      //   rgb.z += (normal.z + 1.0) / 2.0;
      // }
    }
  }

  out[y * w + x] = rgb / (float) spp;
}

Image render_normals(Triangle tri, Camera cam, unsigned int w, unsigned int h) {
  Image out(w, h);

  float3* device_out;
  hipMalloc(&device_out, w * h * sizeof(float3));
  cudaCheckError();

  dim3 block(16, 16);
  dim3 grid((w + 15) / 16, (h + 15) / 16);
  render_kernel_normals<<<grid, block>>>(tri, cam, device_out, w, h);
  cudaCheckError();

  hipMemcpy(out.values.data, device_out, w * h * sizeof(float3), hipMemcpyDeviceToHost);
  cudaCheckError();

  hipFree(device_out);
  cudaCheckError();

  return out;
}

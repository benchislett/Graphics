#include "hip/hip_runtime.h"
#include "bvh.cuh"
#include "camera.cuh"
#include "image.cuh"
#include "render.cuh"
#include "scoped_timer.cuh"
#include "sphere.cuh"
#include "tri_array.cuh"

#include <chrono>
#include <hip/hip_runtime.h>
#include <functional>
#include <iostream>
#include <thread>

#define cudaCheckError()                                                               \
  {                                                                                    \
    hipError_t e = hipGetLastError();                                                \
    if (e != hipSuccess) {                                                            \
      printf("Cuda failure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e)); \
      exit(0);                                                                         \
    }                                                                                  \
  }

struct Path {
  Ray cur;
  float3 L;
  int px;
  int py;
  bool active;
};

__global__ void init_paths(Vector<Path> pq, Camera cam, unsigned int w, unsigned int h, unsigned int spp,
                           unsigned int paths_processed) {
  unsigned int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

  int pp = paths_processed + idx;

  if (idx >= pq.size)
    return;

  int spp_rt = (int) sqrtf((float) spp);

  int sample = pp / (w * h);
  int coord  = pp % (w * h);
  int x      = coord % w;
  int y      = coord / w;
  int sx     = sample % spp_rt;
  int sy     = sample / spp_rt;

  float u = ((float) x + (float) sx / (float) spp_rt) / (float) w;
  float v = ((float) y + (float) sy / (float) spp_rt) / (float) h;

  Ray r = cam.get_ray(u, v);

  pq[idx] = {r, {0, 0, 0}, x, y, true};
}

__global__ void advance_paths(BVH bvh, Vector<Path> pq, Image out, float spp) {
  unsigned int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (idx >= pq.size)
    return;

  Path p = pq[idx];
  Ray r  = p.cur;

  auto i = bvh.intersects(r);

  if (i.hit) {
    pq[idx].cur = Ray(i.point, i.normal);
    Vec3 normal = i.normal;
    // // p.L += normal;
    p.L.x += (normal.x + 1.0) / 2.0;
    p.L.y += (normal.y + 1.0) / 2.0;
    p.L.z += (normal.z + 1.0) / 2.0;
  }

  out[p.py * out.width + p.px] += p.L / spp;
}

Image render_normals(TriangleArray tris, Camera cam, unsigned int w, unsigned int h) {

  Image out(w, h);

  for (int i = 0; i < w * h; i++) {
    out[i] = {0, 0, 0};
  }

  BVH bvh(tris);

  ScopedMicroTimer x_([&](int us) { printf("Rendered in %.2f ms\n", (double) us / 1000.0); });

  unsigned int spp = 1;

  unsigned int total_paths = w * h * spp;

  unsigned int path_queue_size = min(total_paths, 1024 * 1024 * 32);
  Vector<Path> path_queue(path_queue_size);

  unsigned int paths_processed = 0, rounds = 0;
  while (paths_processed < total_paths) {
    dim3 block(128);
    dim3 grid((path_queue_size + 127) / 128);
    init_paths<<<grid, block>>>(path_queue, cam, w, h, spp, paths_processed);
    hipDeviceSynchronize();
    cudaCheckError();
    // for (int idx = 0; idx < path_queue_size; idx++) {
    //   Path p = path_queue[idx];
    //   Ray r  = p.cur;

    //   auto i = bvh.intersects(r);

    //   if (i.hit) {
    //     Vec3 normal = i.normal;
    //     // // p.L += normal;
    //     p.L.x += (normal.x + 1.0) / 2.0;
    //     p.L.y += (normal.y + 1.0) / 2.0;
    //     p.L.z += (normal.z + 1.0) / 2.0;
    //   }

    //   out[p.py * out.width + p.px] += p.L / spp;
    // }
    advance_paths<<<grid, block>>>(bvh, path_queue, out, (float) spp);
    hipDeviceSynchronize();
    cudaCheckError();

    rounds++;
    paths_processed += path_queue_size;
  }

  return out;
}

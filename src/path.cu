#include "hip/hip_runtime.h"
#include "path.cuh"
#include "intersection.cuh"
#include "random.cuh"

__device__ Vec3 sample_li(const Intersection &i, const Primitive &prim, const Scene &s, Vec3 *wi, float u, float v, float *pdf, bool *vis) {
  Vec3 light_p;
  Vec3 light_n;
  prim.t.sample(u, v, pdf, &light_p, &light_n);

  if (*pdf == 0.f || light_p == i.p) {
    *pdf = 0.f;
    *vis = false;
    return Vec3(0.f, 0.f, 0.f);
  }
  *wi = normalized(light_p - i.p);
  *pdf *= length_sq(light_p - i.p) / dot_abs(*wi, light_n);
  *vis = hit_first(Ray(i.p, (*wi)), s, &prim);

  // record visibility
  return s.materials[prim.bsdf].emittance();
}

__device__ float power_heuristic(float nf, float f_pdf, float ng, float g_pdf) {
  float f = nf * f_pdf;
  float g = ng * g_pdf;
  return (f * f) / (f * f + g * g);
}

__device__ Vec3 direct_lighting(const Intersection &i, const Primitive &light, const Scene &s, float u_scatter, float v_scatter, float u_light, float v_light, int bxdf_choice) {
  Vec3 ld = {0.f, 0.f, 0.f};
  Vec3 wi;
  float light_pdf = 0.f, scatter_pdf = 0.f;
  bool visible;

  BSDF& mat = s.materials[i.prim->bsdf];

  Vec3 f;
  float weight;

  // Sample light
  Vec3 li = sample_li(i, light, s, &wi, u_light, v_light, &light_pdf, &visible);
  if (light_pdf != 0.f && !is_zero(li)) {
    f = mat.f(i.incoming, wi) * dot_abs(wi, i.n);
    scatter_pdf = mat.pdf(i.incoming, wi);

    if (!is_zero(f)) {
      if (!visible) li = {0.f, 0.f, 0.f};

      if (!is_zero(li)) {
        weight = power_heuristic(1.f, light_pdf, 1.f, scatter_pdf);
        ld += f * li * weight / light_pdf;
      }
    }
  }

  // Sample BSDF
  /*
  f = mat.sample_f(i.incoming, &wi, u_scatter, v_scatter, &scatter_pdf, bxdf_choice);
  f *= dot_abs(wi, i.n);
  if (!is_zero(f) && scatter_pdf != 0.f) {
    weight = power_heuristic(1.f, scatter_pdf, 1.f, light_pdf);
    bool did_hit = hit_first(Ray(i.p, wi), s, &light);
    if (did_hit) ld += f * li * weight / scatter_pdf;
  }*/

  return ld;
}

__device__ Vec3 sample_one_light(const Intersection &i, const Scene &s, float u_scatter, float v_scatter, float u_light, float v_light, int light_idx, int bxdf_idx) {
  if (s.lights.size() == 0) return Vec3(0.f);

  return direct_lighting(i, s.prims[s.lights[light_idx]], s, u_scatter, v_scatter, u_light, v_light, bxdf_idx) * (float)s.lights.size();
}

__device__ Vec3 trace(const Ray &r, const Scene &scene, LocalDeviceRNG &gen, int max_depth) {
  Vec3 l = {0.f, 0.f, 0.f};
  Vec3 beta = {1.f, 1.f, 1.f};
  Ray ray = r;
  Vec3 wo_world, wi_world;
  float pdf;
  Vec3 f;
  Vec3 uvw;

  bool does_hit;
  Intersection i;
  int bounces;
  for (bounces = 0;; bounces++) {
    does_hit = hit(ray, scene, &i);

    if (!does_hit || bounces >= max_depth) {
      // l = beta * Vec3(1.f, 1.f, 1.f);
      break;
    }

    BSDF &mat = scene.materials[i.prim->bsdf];

    uvw = {i.u, i.v, 1.f - i.u - i.v};
    uvw = (i.prim->t.t_a * uvw.e[2]) + (i.prim->t.t_b * uvw.e[0]) + (i.prim->t.t_c * uvw.e[1]);
    mat.update(i.n, i.s, scene.textures, uvw.e[0], uvw.e[1]);
    
    if (mat.is_light() && bounces == 0) {
      l += beta * mat.emittance();
      break;
    }

    int n = mat.n_bxdfs;
    int light_choice = gen.generate_int(0, scene.lights.size() - 1);
    int bxdf_choice = (n == 1) ? 0 : gen.generate_int(0, n - 1);
    l += beta * sample_one_light(i, scene, gen.generate(), gen.generate(), gen.generate(), gen.generate(), light_choice, bxdf_choice);

    wo_world = i.incoming;
    int choice = (n == 1) ? 0 : gen.generate_int(0, n - 1);
    f = mat.sample_f(wo_world, &wi_world, gen.generate(), gen.generate(), &pdf, choice);

    if (is_zero(f) || fabsf(pdf) < 0.0001f) break;

    float cos_term = dot_abs(wi_world, i.n);
    beta *= f * cos_term / pdf;
    ray = Ray(i.p, wi_world);
  }
  return l;
}

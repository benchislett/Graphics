#include "hip/hip_runtime.h"
#include "image.cuh"
#include "lodepng.h"

#include <iostream>
#include <vector>

Image::Image(const std::string& filename) : values{width * height} {
  std::vector<unsigned char> pixels;

  unsigned error = lodepng::decode(pixels, width, height, filename);
  if (error) {
    std::cerr << "decoder error " << error << ": " << lodepng_error_text(error) << std::endl;
  }

  for (unsigned int i = 0; i < width * height * 4; i += 4) {
    float r = ((float) pixels[i + 0]) / 255.0;
    float g = ((float) pixels[i + 1]) / 255.0;
    float b = ((float) pixels[i + 2]) / 255.0;
    // ignore alpha

    values[i / 4] = {r, g, b};
  }
}

void Image::to_png(const std::string& filename) const {
  std::vector<unsigned char> pixels(width * height * 4);

  for (unsigned int i = 0; i < width * height * 4; i += 4) {
    float3 rgb    = values[i / 4];
    pixels[i + 0] = (unsigned char) (255.0 * rgb.x);
    pixels[i + 1] = (unsigned char) (255.0 * rgb.y);
    pixels[i + 2] = (unsigned char) (255.0 * rgb.z);
    pixels[i + 3] = 255;
  }

  unsigned error = lodepng::encode(filename, pixels, width, height);
  if (error) {
    std::cerr << "encoder error " << error << ": " << lodepng_error_text(error) << std::endl;
  }
}
